// CUDA Program to add two matrices


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<chrono>

#define MAX_SIZE 5000
#define TRUE 1
#define FALSE 0
int GRID_SIZE;        // Total Number of blocks
int BLOCK_SIZE;       // Total Number of threads in one block

void InitMatrix(double *array, int is_empty);
int Allocate2DMemory(double ***array, int n, int m);
int Allocate2DMemoryDevice(double ***array, int n, int m);
int Free2DMemory(double ***array);
int Free2DMemoryDevice(double ***array);
void PrintMatrix(double * array, char * array_name);
void TestSum(double *A, double *B, int len);



__global__ void matrix_add(double *a, double *b, double *sum, int *dev_block_size, int *dev_grid_size) {
    long long idx = (blockIdx.x)*(*dev_block_size) + threadIdx.x;
    int num_procs = (*dev_block_size)*(*dev_grid_size);
    int len = MAX_SIZE/num_procs;
    if((long long)(len*idx + len - 1) < (long long)MAX_SIZE) {
        for(long long i = 0; i < len; i++)
            for(long long j = 0; j < MAX_SIZE; j++)
                sum[(i + len*idx)*MAX_SIZE + j] = a[(i + len*idx)*MAX_SIZE + j] + b[(i + len*idx)*MAX_SIZE + j];
    }
    if(blockIdx.x * threadIdx.x == 0) {
        for(long long i = 0; i < (MAX_SIZE%num_procs); i++) {
            for(long long j = 0; j < MAX_SIZE; j++)
                sum[(MAX_SIZE-1-i)*MAX_SIZE + j] = a[(MAX_SIZE-1-i)*MAX_SIZE + j] + b[(MAX_SIZE-1-i)*MAX_SIZE + j];
        }
    }
}



int main(void) {
    printf("Enter the grid size and the block size respectively:\n");
    scanf("%d %d", &GRID_SIZE, &BLOCK_SIZE);
    int size = sizeof(double) * MAX_SIZE * MAX_SIZE;
    
    // Initialising the matrices
    double *matrix1 = (double *)malloc(size);
    double *matrix2 = (double *)malloc(size);
    double *final_sum_matrix = (double *)malloc(size);
    srand(time(0));
    InitMatrix(matrix1, FALSE);
    InitMatrix(matrix2, FALSE);
    
    auto start = std::chrono::high_resolution_clock::now();
    // copying the data to the device
    double *dev_m1;
    double *dev_m2;
    double *dev_sum;
    int *dev_block_size, *dev_grid_size;
    hipMalloc((void **)&dev_m1, size);
    hipMalloc((void **)&dev_m2, size);
    hipMalloc((void **)&dev_block_size, sizeof(int));
    hipMalloc((void **)&dev_grid_size, sizeof(int));
    hipMalloc((void **)&dev_sum, size);
    hipMemcpy(dev_m1, matrix1, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_m2, matrix2, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_block_size, &BLOCK_SIZE, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_grid_size, &GRID_SIZE, sizeof(int), hipMemcpyHostToDevice);

    matrix_add<<<GRID_SIZE,BLOCK_SIZE>>>(dev_m1, dev_m2, dev_sum, dev_block_size, dev_grid_size);
    hipError_t err = hipMemcpy(final_sum_matrix, dev_sum, size, hipMemcpyDeviceToHost);
	if(err != hipSuccess) {
		printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
	}

    // PrintMatrix(matrix1, "A: ");  // uncomment to display the A matrix
    // PrintMatrix(matrix2, "B: ");  // uncomment to display the B matrix
    // PrintMatrix(final_sum_matrix, "Sum: "); // uncomment to display the final sum matrix
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    printf("Time Taken: %ld\n", duration.count());
    // TestSum(matrix1, matrix2, MAX_SIZE); // uncomment to check the actual sum without using CUDA

    // Cleanup
    free(matrix1);
    free(matrix2);
    free(final_sum_matrix);
    hipFree(dev_m1);
    hipFree(dev_m2);
    hipFree(dev_sum);
    return 0;

}


// Initializes the 2D matrix
void InitMatrix(double *matrix, int is_empty) {
    for(int i = 0; i < MAX_SIZE; i++) {
        for(int j = 0; j < MAX_SIZE; j++) {
            if(is_empty)
                matrix[i*MAX_SIZE + j] = -1;
            else    
                matrix[i*MAX_SIZE + j] = (double)(rand() % 100000) / (double)100;
        }
   } 
}

// prints the matrix 
void PrintMatrix(double * matrix, char * matrix_name) {
    printf("The Matrix %s:\n", matrix_name);
    for(int i = 0; i < MAX_SIZE; i++) {
        for(int j = 0; j < MAX_SIZE; j++) {
            printf("%f ", matrix[i*MAX_SIZE + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// Calculates sum without using CUDA for testing purpose
void TestSum(double *A, double *B, int len) {
    printf("This output is without using CUDA : \n");
    for(int i = 0; i < len; i++) {
        for(int j = 0; j < len; j++)
            printf("%f ", A[i*len + j] + B[i*len + j]);
        printf("\n");
    }
    printf("\n\n");
}